// nvcc -o laplacian laplacian.cu -std=c++11
// nvcc -o laplacian laplacian.cu -std=c++11 -O3 -g -D_FORCE_INLINES

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdlib>
#include <chrono>
#include <cmath>
#include <cstring>

__global__
void gpu_laplacian(unsigned int width, unsigned int height, float* values, float* laplacian) {
}

__host__
void cpu_laplacian(unsigned int width, unsigned int height, float* values, float* laplacian) {

  // For efficiently computing the laplacian we will move 5 pointers on values
  // pointing : on the pixel, on its north, on its east, on its west, on its south
  float* v_ptr, *vN_ptr, *vE_ptr, *vS_ptr, *vW_ptr;
  float* l_ptr;

  // We handle the corners
  // #oooooo#
  // oooooooo
  // oooooooo
  // oooooooo
  // #oooooo#
  laplacian[0] = -(4 * values[0]) + (values[(height-1)*width] + values[1] + values[width] + values[width-1]);// top left
  laplacian[width-1] = -(4 * values[width-1]) + (values[(height-1)*width + (width-1)] + values[0] + values[width + (width-1)] + values[width-2]); // top right
  laplacian[(height-1)*width] = -(4 * values[(height-1)*width]) + (values[(height-2)*width] + values[(height-1)*width + 1] + values[0] + values[(height-1)*width + (width-1)]); // bottom left
  laplacian[(height-1)*width + (width-1)] = -(4 * values[(height-1)*width + (width-1)]) + (values[(height-2)*width + (width-1)]+values[(height-1)*width] + values[width-1] + values[(height-1)*width + (width-2)]); // bottom right

  // We handle the borders
  // o########o
  // oooooooooo
  // oooooooooo
  // oooooooooo
  // oooooooooo
  v_ptr = values + 1;
  vN_ptr = values + (height-1)*width + 1;
  vE_ptr = values + 2;
  vS_ptr = values + width + 1;
  vW_ptr = values;
  l_ptr = laplacian + 1;
  for(unsigned int i = 1 ; i < width-1 ; ++i, ++v_ptr, ++vN_ptr, ++vE_ptr, ++vS_ptr, ++vW_ptr, ++l_ptr) 
    *l_ptr = (*vN_ptr + *vE_ptr + *vS_ptr + *vW_ptr) - ((*v_ptr) * 4);

  // oooooooooo
  // oooooooooo
  // oooooooooo
  // oooooooooo
  // o########o
  v_ptr = values + (height-1)*width + 1;
  vN_ptr = values + (height-2)*width + 1;
  vE_ptr = values + (height-1)*width + 2;
  vS_ptr = values + 1 ;
  vW_ptr = values + (height-1)*width;
  l_ptr = laplacian + (height-1)*width + 1;
  for(unsigned int i = 1 ; i < width-1 ; ++i, ++v_ptr, ++vN_ptr, ++vE_ptr, ++vS_ptr, ++vW_ptr, ++l_ptr) 
    *l_ptr = (*vN_ptr + *vE_ptr + *vS_ptr + *vW_ptr) - ((*v_ptr) * 4);

  // oooooooooo
  // #ooooooooo
  // #ooooooooo
  // #ooooooooo
  // #ooooooooo
  // oooooooooo
  v_ptr = values + width;
  vN_ptr = values ;
  vE_ptr = values + width + 1;
  vS_ptr = values + 2*width ;
  vW_ptr = values + width + (width-1);
  l_ptr = laplacian + width;
  for(unsigned int i = 1 ; i < height-1 ; ++i, v_ptr+=width, vN_ptr+=width, vE_ptr+=width, vS_ptr+=width, vW_ptr+=width, l_ptr+=width) 
    *l_ptr = (*vN_ptr + *vE_ptr + *vS_ptr + *vW_ptr) - ((*v_ptr) * 4);

 
  // oooooooooo
  // ooooooooo#
  // ooooooooo#
  // ooooooooo#
  // ooooooooo#
  // oooooooooo
  // 
  v_ptr = values + width + (width-1);
  vN_ptr = values + (width-1);
  vE_ptr = values + width;
  vS_ptr = values + 2*width + (width-1) ;
  vW_ptr = values + width + (width-2);
  l_ptr = laplacian + width + (width-1);
  for(unsigned int i = 1 ; i < height-1 ; ++i, v_ptr+=width, vN_ptr+=width, vE_ptr+=width, vS_ptr+=width, vW_ptr+=width, l_ptr+=width) 
    *l_ptr = (*vN_ptr + *vE_ptr + *vS_ptr + *vW_ptr) - ((*v_ptr) * 4);

  // We handle the region inside the array exlucding a border of size 1,
  // i.e. the pixels # below
  // oooooooooo
  // o########o
  // o########o
  // o########o
  // oooooooooo
  v_ptr = values + (1*width + 1);
  vN_ptr = values + 1;
  vE_ptr = values + (1*width + 2);
  vS_ptr = values + (2*width + 1);
  vW_ptr = values + (1*width + 0);

  l_ptr = laplacian + (1*width + 1);
  for(unsigned int i = 1; i < height-1; ++i) {
    for(unsigned int j = 1 ; j < width-1 ; ++j, ++vN_ptr, ++vE_ptr, ++vS_ptr, ++vW_ptr, ++v_ptr, ++l_ptr) 
      *l_ptr = (*vN_ptr + *vE_ptr + *vS_ptr + *vW_ptr) - ((*v_ptr) * 4);

    // For switching to the next line we must move the pointers forward by 2 pixels
    v_ptr += 2;
    l_ptr += 2;
    vN_ptr += 2;
    vE_ptr += 2;
    vS_ptr += 2;
    vW_ptr += 2;
  }
} 

__host__
float diffNorm(float* v1, float* v2, unsigned int N) {
  float res = 0.0;
  float* v1ptr = v1;
  float* v2ptr = v2;
  float d;
  for(unsigned int i = 0 ; i < N*N; ++i, ++v1ptr, ++v2ptr) {
    d = (*v1ptr) - (*v2ptr);
    res += d*d;
  }
  return sqrt(d);
}

__host__
int main(int argc, char * argv[]) {

  unsigned int width = 256;
  unsigned int height = 256;
  unsigned int nbcalls = 500;

  float *I, *lcpu, *lgpu, *dI, *dlgpu;
  I = (float*) malloc(width*height*sizeof(float));
  lcpu = (float*) malloc(width*height*sizeof(float));
  lgpu = (float*) malloc(width*height*sizeof(float));

  // Initialize the input image
  float *Iptr = I;
  for(unsigned int i = 0 ; i < width*height; ++i, ++Iptr)
    (*Iptr) = std::rand() / ((float)RAND_MAX);

  //************* CPU *****************//
  std::chrono::time_point<std::chrono::system_clock> start_cpu, end_cpu;
  start_cpu = std::chrono::system_clock::now();

  for(unsigned int i = 0 ; i < nbcalls; ++i) 
    cpu_laplacian(width, height, I, lcpu);
  
  end_cpu = std::chrono::system_clock::now();
  int elapsed_cpu_ms = std::chrono::duration_cast<std::chrono::milliseconds>(end_cpu-start_cpu).count();
  printf("CPU elapsed : %f ms per call \n", ((float)elapsed_cpu_ms)/nbcalls);

  //************* GPU *****************//
  
  size_t pitch_dI, pitch_dlgpu;
  hipMallocPitch(&dI, &pitch_dI, width*sizeof(float), height);
  hipMallocPitch(&dlgpu, &pitch_dlgpu, width*sizeof(float), height);
  
  std::chrono::time_point<std::chrono::system_clock> start_gpu, end_gpu;
  start_gpu = std::chrono::system_clock::now();
  
  // Copy the input to the GPU
  //cudaMemcpy(dI, I, N*N*sizeof(float), cudaMemcpyHostToDevice);
  hipMemcpy2D(dI, pitch_dI, I, width*sizeof(float), width*sizeof(float), height, hipMemcpyHostToDevice);
  
  // Call the kernel
  int threadsPerBlock = 256;
  int blocksPerGrid = 1;
  gpu_laplacian<<<blocksPerGrid, threadsPerBlock>>>(width, height, dI, dlgpu);
  
  // Get the result
  //cudaMemcpy(lgpu, dlgpu, N*N*sizeof(float), cudaMemcpyDeviceToHost);
  hipMemcpy2D(lgpu, width*sizeof(float), dlgpu, width*sizeof(float), width*sizeof(float), height, hipMemcpyDeviceToHost);
  
  end_gpu = std::chrono::system_clock::now();
  int elapsed_gpu_ms = std::chrono::duration_cast<std::chrono::milliseconds>(end_gpu-start_gpu).count();
  printf("GPU elapsed : %f ms per call \n", ((float)elapsed_gpu_ms)/nbcalls);
  
  //********** Comparison *************//
  printf("Difference : %f \n", diffNorm(lcpu, lgpu, width*height));
  
  //***********************************//
  // Free the device memory
  hipFree(dI);
  hipFree(dlgpu);

  // Free the host memory
  delete[] I;
  delete[] lcpu;
  delete[] lgpu;
}
